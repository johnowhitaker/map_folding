#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <thread>
#include <hip/hip_runtime.h>

typedef unsigned long long ull;
const int MAX_N = 64;
const int MAX_GAP = MAX_N * MAX_N + 1;

__constant__ int bigP_const[3];
__constant__ int c_const[3][MAX_N + 1];
__constant__ int d_const[3][MAX_N + 1][MAX_N + 1];

struct State {
    int a[MAX_N + 1];
    int b_arr[MAX_N + 1];
    int gapter[MAX_N + 2];
};

std::vector<State> collect_partial_states(int grid_size, int partition_depth, int hc[3][MAX_N + 1], int hd[3][MAX_N + 1][MAX_N + 1]) {
    int dim = 2;
    int p[2] = {grid_size, grid_size};
    int n = p[0] * p[1];

    std::vector<State> states;
    states.reserve(100000);

    int *a = new int[n + 1];
    int *b_arr = new int[n + 1];
    int *gapter = new int[n + 2];
    int *count = new int[n + 1];
    int *gap = new int[MAX_GAP];

    memset(a, 0, (n + 1) * sizeof(int));
    memset(b_arr, 0, (n + 1) * sizeof(int));
    memset(gapter, 0, (n + 2) * sizeof(int));
    memset(count, 0, (n + 1) * sizeof(int));
    memset(gap, 0, MAX_GAP * sizeof(int));

    b_arr[0] = 1; // Sentinel

    int flag = 1;
    int res = 0;
    int mod = 0;
    int g = 0;
    int l_leaf = 1;
    gapter[0] = 0;

    while (l_leaf > 0) {
        if (!flag || l_leaf <= 1 || b_arr[0] == 1) {
            if (l_leaf > n) {
                // Should not reach here
            } else if (l_leaf > partition_depth) {
                // Collect the partial state and force backtrack without computing gaps
                State s;
                memcpy(s.a, a, (n + 1) * sizeof(int));
                memcpy(s.b_arr, b_arr, (n + 1) * sizeof(int));
                memcpy(s.gapter, gapter, (n + 2) * sizeof(int));
                states.push_back(s);
                // No need to set g; it's already set to trigger pop in the while loop
            } else {
                int dd = 0;
                int gg = gapter[l_leaf - 1];
                g = gg;

                for (int i = 1; i <= dim; i++) {
                    if (hd[i][l_leaf][l_leaf] == l_leaf) {
                        dd++;
                    } else {
                        int m = hd[i][l_leaf][l_leaf];
                        while (m != l_leaf) {
                            if (mod == 0 || l_leaf != mod || m % mod == res) {
                                gap[gg] = m;
                                if (count[m]++ == 0) gg++;
                            }
                            m = hd[i][l_leaf][b_arr[m]];
                        }
                    }
                }

                if (dd == dim) {
                    for (int m = 0; m < l_leaf; m++) {
                        gap[gg++] = m;
                    }
                }

                for (int j = g; j < gg; j++) {
                    gap[g] = gap[j];
                    if (count[gap[j]] == dim - dd) {
                        g++;
                    }
                    count[gap[j]] = 0;
                }
            }
        }

        while (l_leaf > 0 && g == gapter[l_leaf - 1]) {
            l_leaf--;
            b_arr[a[l_leaf]] = b_arr[l_leaf];
            a[b_arr[l_leaf]] = a[l_leaf];
        }

        if (l_leaf > 0) {
            a[l_leaf] = gap[--g];
            b_arr[l_leaf] = b_arr[a[l_leaf]];
            b_arr[a[l_leaf]] = l_leaf;
            a[b_arr[l_leaf]] = l_leaf;
            gapter[l_leaf] = g;
            l_leaf++;
        }
    }

    delete [] a;
    delete [] b_arr;
    delete [] gapter;
    delete [] count;
    delete [] gap;

    return states;
}

__global__ void compute_counts(int grid_size, int partition_depth, int num_states, int *all_a, int *all_b_arr, int *all_gapter, ull *result) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= num_states) return;

    int n = grid_size * grid_size;
    int dim = 2;
    int p[2] = {grid_size, grid_size};

    int a[MAX_N + 1];
    int b_arr[MAX_N + 1];
    int gapter[MAX_N + 2];
    int count[MAX_N + 1] = {0};
    int gap[MAX_GAP] = {0};

    for (int i = 0; i <= n; i++) {
        a[i] = all_a[id * (n + 1) + i];
        b_arr[i] = all_b_arr[id * (n + 1) + i];
    }
    for (int i = 0; i <= n + 1; i++) {
        gapter[i] = all_gapter[id * (n + 2) + i];
    }

    int flag = 1;
    int res = 0;
    int mod = 0;
    int g = 0;
    int l_leaf = partition_depth + 1;
    ull myCount = 0;

    while (l_leaf > 0) {
        if (!flag || l_leaf <= 1 || b_arr[0] == 1) {
            if (l_leaf > n) {
                myCount += n;
            } else {
                int dd = 0;
                int gg = gapter[l_leaf - 1];
                g = gg;

                for (int i = 1; i <= dim; i++) {
                    if (d_const[i][l_leaf][l_leaf] == l_leaf) {
                        dd++;
                    } else {
                        int m = d_const[i][l_leaf][l_leaf];
                        while (m != l_leaf) {
                            if (mod == 0 || l_leaf != mod || m % mod == res) {
                                gap[gg] = m;
                                if (count[m]++ == 0) gg++;
                            }
                            m = d_const[i][l_leaf][b_arr[m]];
                        }
                    }
                }

                if (dd == dim) {
                    for (int m = 0; m < l_leaf; m++) {
                        gap[gg++] = m;
                    }
                }

                for (int j = g; j < gg; j++) {
                    gap[g] = gap[j];
                    if (count[gap[j]] == dim - dd) {
                        g++;
                    }
                    count[gap[j]] = 0;
                }
            }
        }

        while (l_leaf > 0 && g == gapter[l_leaf - 1]) {
            l_leaf--;
            b_arr[a[l_leaf]] = b_arr[l_leaf];
            a[b_arr[l_leaf]] = a[l_leaf];
        }

        if (l_leaf > 0) {
            a[l_leaf] = gap[--g ];
            b_arr[l_leaf] = b_arr[a[l_leaf]];
            b_arr[a[l_leaf]] = l_leaf;
            a[b_arr[l_leaf]] = l_leaf;
            gapter[l_leaf] = g;
            l_leaf++;
        }
    }

    atomicAdd(result, myCount);
}

void process_gpu(int device_id, int grid_size, int partition_depth, int start_idx, int end_idx, const std::vector<State>& states, ull& partial_result) {
    hipSetDevice(device_id);

    int num_local_states = end_idx - start_idx;
    if (num_local_states <= 0) {
        partial_result = 0;
        return;
    }

    int n = grid_size * grid_size;

    int *d_all_a, *d_all_b_arr, *d_all_gapter;
    ull *d_result;
    hipMalloc(&d_all_a, num_local_states * (n + 1) * sizeof(int));
    hipMalloc(&d_all_b_arr, num_local_states * (n + 1) * sizeof(int));
    hipMalloc(&d_all_gapter, num_local_states * (n + 2) * sizeof(int));
    hipMalloc(&d_result, sizeof(ull));

    int *h_all_a = new int[num_local_states * (n + 1)];
    int *h_all_b_arr = new int[num_local_states * (n + 1)];
    int *h_all_gapter = new int[num_local_states * (n + 2)];

    for (int s = 0; s < num_local_states; s++) {
        const State& state = states[start_idx + s];
        memcpy(h_all_a + s * (n + 1), state.a, (n + 1) * sizeof(int));
        memcpy(h_all_b_arr + s * (n + 1), state.b_arr, (n + 1) * sizeof(int));
        memcpy(h_all_gapter + s * (n + 2), state.gapter, (n + 2) * sizeof(int));
    }

    hipMemcpy(d_all_a, h_all_a, num_local_states * (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_all_b_arr, h_all_b_arr, num_local_states * (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_all_gapter, h_all_gapter, num_local_states * (n + 2) * sizeof(int), hipMemcpyHostToDevice);

    ull h_result = 0;
    hipMemcpy(d_result, &h_result, sizeof(ull), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_blocks = (num_local_states + block_size - 1) / block_size;
    compute_counts<<<grid_blocks, block_size>>>(grid_size, partition_depth, num_local_states, d_all_a, d_all_b_arr, d_all_gapter, d_result);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(ull), hipMemcpyDeviceToHost);
    partial_result = h_result;

    delete[] h_all_a;
    delete[] h_all_b_arr;
    delete[] h_all_gapter;
    hipFree(d_all_a);
    hipFree(d_all_b_arr);
    hipFree(d_all_gapter);
    hipFree(d_result);
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: ./gpu_folding grid_size partition_depth\n");
        printf("Example: ./gpu_folding 8 20\n");
        return 1;
    }

    int grid_size = atoi(argv[1]);
    int partition_depth = atoi(argv[2]);
    int n = grid_size * grid_size;
    if (n > MAX_N) {
        printf("Grid too large, max 8 (64 stamps)\n");
        return 1;
    }

    int dim = 2;
    int p[2] = {grid_size, grid_size};

    int hbigP[3];
    hbigP[0] = 1;
    for (int i = 1; i <= dim; i++) {
        hbigP[i] = hbigP[i - 1] * p[i - 1];
    }

    int hc[3][MAX_N + 1];
    for (int i = 1; i <= dim; i++) {
        for (int m = 1; m <= n; m++) {
            hc[i][m] = (m - 1) / hbigP[i - 1] - ((m - 1) / hbigP[i]) * p[i - 1] + 1;
        }
    }

    int hd[3][MAX_N + 1][MAX_N + 1];
    for (int i = 1; i <= dim; i++) {
        for (int l = 1; l <= n; l++) {
            for (int m = 1; m <= l; m++) {
                int delta = hc[i][l] - hc[i][m];
                if ((delta & 1) == 0) {
                    if (hc[i][m] == 1) {
                        hd[i][l][m] = m;
                    } else {
                        hd[i][l][m] = m - hbigP[i - 1];
                    }
                } else {
                    if (hc[i][m] == p[i - 1] || m + hbigP[i - 1] > l) {
                        hd[i][l][m] = m;
                    } else {
                        hd[i][l][m] = m + hbigP[i - 1];
                    }
                }
            }
        }
    }

    auto states = collect_partial_states(grid_size, partition_depth, hc, hd);
    int num_states = states.size();
    printf("Number of partial states at depth %d: %d\n", partition_depth, num_states);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    printf("Detected %d GPUs\n", num_gpus);

    for (int dev = 0; dev < num_gpus; ++dev) {
        hipSetDevice(dev);
        hipMemcpyToSymbol(HIP_SYMBOL(bigP_const), hbigP, sizeof(hbigP));
        hipMemcpyToSymbol(HIP_SYMBOL(c_const), hc, sizeof(hc));
        hipMemcpyToSymbol(HIP_SYMBOL(d_const), hd, sizeof(hd));
    }

    std::vector<ull> partial_results(num_gpus, 0);
    std::vector<std::thread> threads;

    int states_per_gpu = num_states / num_gpus;
    int remainder = num_states % num_gpus;

    int current_start = 0;
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        int local_num = states_per_gpu + (gpu < remainder ? 1 : 0);
        int end_idx = current_start + local_num;

        threads.emplace_back([gpu, grid_size, partition_depth, current_start, end_idx, &states, &partial_results]() {
            ull local_result;
            process_gpu(gpu, grid_size, partition_depth, current_start, end_idx, states, local_result);
            partial_results[gpu] = local_result;
        });

        current_start = end_idx;
    }

    for (auto& t : threads) {
        t.join();
    }

    ull total_result = 0;
    for (auto res : partial_results) {
        total_result += res;
    }

    printf("Total number of ways: %llu\n", total_result);

    return 0;
}